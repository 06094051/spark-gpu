
#include <hip/hip_runtime.h>
__global__ void multiplyBy2(int *in, int *out, long size) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < size) {
        out[ix] = in[ix] * 2;
    }
}
